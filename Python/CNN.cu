
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

void charBckgrndPrint(char *str, float rgb){
  printf("\033[48;2;%d;%d;%dm", (int) rgb*255, (int) rgb*255, (int) rgb*255);
  printf("%s\033[0m",str);
}

void imgColorPrint(int height, int width, float *img){
  int row, col;
  char *str="  ";
  for(row=0; row<height; row++){
    for(col=0; col<width; col++){
      charBckgrndPrint(str,img[row*width + col]);
    }
    printf("\n");
  }
}

void Matrix2DInitRand(float *M, int n, int p)
{
    srand(time(0));
    for(int i=0; i<n; i++) {
      for(int j=0;j<p;j++) {
         M[i*n + j] = (float)rand() / (float)RAND_MAX ;
      }
    }
}

void Matrix3DInitZero(float *M, int fm, int n, int p)
{
    for(int i=0; i<n; i++) {
      for(int j=0;j<p;j++) {
        for(int k=0;k<fm;k++) {
          M[i*p + j*fm + k] = 0;
        }
      }
    }
}

void Matrix3DInitRand(float *M, int fm, int n, int p)
{
  srand(time(0));
  for(int i=0; i<n; i++) {
    for(int j=0;j<p;j++) {
      for(int k=0;k<fm;k++) {
        M[i*p + j*fm + k] = (float)rand() / (float)RAND_MAX;
      }
    }
  }
}

void Matrix2DPrint(float *M, int n, int p)
{
    for(int i=0; i<n; i++) {
      for(int j=0;j<p;j++) {
          printf("%f |", M[i*n + j] );
      }
      printf ( "\n");
    }
}


void activation_softmax(float *vin, int n, float *vout)
{
  float sum = 0;
  for(int i=0; i<n; i++) {
    sum += vin[i];
  }
  for(int i=0; i<n; i++) {
    vout[i] = vin[i]/sum;
  }
}

__global__ void cudaConv2D(float* M, float* kernel, float* Mout, int M_ligne, int M_colonne, int kernel_size, int nb_kernel, int Mout_ligne, int Mout_colonne){
    
    //Convolution d'une matrice par un kernel
    int lig = blockIdx.x;
    int col = threadIdx.x;

    float s = 0.0;

    if (lig < Mout_ligne && col < Mout_colonne)
    {
        int tot = M_ligne * M_colonne;

        for (int kernel_lig = 0; kernel_lig < kernel_size; kernel_lig++) {
            for (int kernel_col = 0; kernel_col < kernel_size; kernel_col++) {
                for (int n_k = 0; n_k < nb_kernel; n_k++)
                {
                    s += M[(lig + kernel_lig) * M_colonne + col + kernel_col + n_k * tot] * kernel[kernel_lig * kernel_size + kernel_col + n_k * nb_kernel];
                }
            }
        }
        Mout[lig * Mout_colonne + col] = s;
    }
}

__global__ void cudaMeanPool(float* M, float* Mout, int M_ligne, int M_colonne, int M_prof, int meanpool_size, int Mout_ligne, int Mout_colonne){
    /*
      cudaMeanPool réalise un sous-échantillonanage par moyennage 2x2 pixels vers 1 pixel.
    */

    // On effectuera les calculs sur les lignes et colonnes paires. 
    // ATTENTION: Dans ce cas blockIdx.x et threadIdx.x vont de 0 à 13 donc lig et col de 0 à 26.
    int lig = 2*blockIdx.x;
    int col = 2*threadIdx.x;


        float s;
        int tot_meanpool = meanpool_size * meanpool_size;
        int tot_M = M_ligne * M_colonne;
        int tot_Mout = Mout_ligne * Mout_colonne;
        
        for (int n_prof = 0; n_prof < M_prof; n_prof++){
            s = 0.0;
            
            for (int meanpool_lig = 0; meanpool_lig < meanpool_size; meanpool_lig++) {
                for (int meanpool_col = 0; meanpool_col < meanpool_size; meanpool_col++) {
                    s += M[(lig + meanpool_lig) * M_colonne + col + meanpool_col + n_prof * tot_M] / tot_meanpool;
                }
            }

            Mout[blockIdx.x * Mout_colonne + threadIdx.x + n_prof * tot_Mout] = s;
            
        }
}

__global__ void activation_tanh(float* M, int M_ligne, int M_colonne, int M_prof, float* Mout){
    
    int lig = blockIdx.x;
    int col = threadIdx.x;

    int tot_M = M_ligne * M_colonne;
    
    for (int n_prof = 0; n_prof < M_prof; n_prof++){
        Mout[lig * M_colonne + col + n_prof * tot_M] = tanh(M[lig * M_colonne + col + n_prof * tot_M]);
    }

}

__global__ void Dense(float *A, float *v, float *vout, int col) {
    int lig = blockIdx.x;
    // Handling arbitrary vector size
    for(int k=0;k<col;k++) {
        vout[lig] += A[lig*col + k] * v[k];
    }
}


int main(int argc, char *argv[]){

  // Layer 1
  // Initialisation de la matrice d'entrée
  float *raw_data;
  raw_data = (float*)malloc(sizeof(float) * 32 * 32);
  Matrix2DInitRand(raw_data, 32, 32);

  //imgColorPrint(32, 32, raw_data);

  // Initialisation de la matrice de sortie de la première conv
  float *C1_data;
  C1_data = (float*)malloc(sizeof(float) * 6 * 28 * 28);
  Matrix3DInitZero(C1_data, 6, 28, 28);
   
  // Initialisation de la matrice de sortie du sous-echantillonnage
  float *S1_data;
  S1_data = (float*)malloc(sizeof(float) * 6 * 14 * 14);
  Matrix3DInitZero(S1_data, 6, 14, 14);
  
  // Initialisation de la matrice des premiers kernels
  float *C1_kernel;
  C1_kernel = (float*)malloc(sizeof(float) * 6 * 5 * 5);
  Matrix3DInitRand(C1_kernel, 6, 5, 5);

  float *S2_data;
  S2_data = (float*)malloc(sizeof(float) * 16 * 5 * 5);
  Matrix3DInitZero(S1_data, 16, 5, 5);

  float *pred;
  pred = (float*)malloc(sizeof(float) * 10);

  // Copie des matrices dans la mémoire GPU afin d'effectuer les calculs du réseau
  
  float *d_raw_data, *d_C1_data, *d_C1_data_activated, *d_C1_kernel, *d_S1_data;
  float *d_C2_data, *d_C2_data_activated, *d_C2_kernel, *d_S2_data;
  float *d_dense1_weigths, *d_dense1, *d_dense1_activated;
  float *d_dense2_weigths, *d_dense2, *d_dense2_activated;
  float *d_dense3_weigths, *d_dense3, *d_pred;

  hipMalloc((void**)&d_raw_data, sizeof(float) * 32 * 32 * 1);
  hipMemcpy(d_raw_data, raw_data, sizeof(float) * 32 * 32 * 1, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_C1_kernel, sizeof(float) * 5 * 5 * 6);
  hipMalloc((void**)&d_C1_data, sizeof(float) * 28 * 28 * 6);
  hipMalloc((void**)&d_C1_data_activated, sizeof(float) * 28 * 28 * 6);
  hipMalloc((void**)&d_S1_data, sizeof(float) * 14 * 14 * 6);
  hipMalloc((void**)&d_C2_data, sizeof(float) * 16 * 10 * 10);
  hipMalloc((void**)&d_C2_data_activated, sizeof(float) * 16 * 10 * 10);
  hipMalloc((void**)&d_C2_kernel, sizeof(float) * 16 * 5 * 5);
  hipMalloc((void**)&d_S2_data, sizeof(float) * 16 * 5 * 5);

  hipMalloc((void**)&d_dense1_weigths, sizeof(float) * 120 * 16 * 5 * 5);
  hipMalloc((void**)&d_dense1, sizeof(float) * 120);
  hipMalloc((void**)&d_dense1_activated, sizeof(float) * 120);
  hipMalloc((void**)&d_dense2_weigths, sizeof(float) * 120 * 84);
  hipMalloc((void**)&d_dense2, sizeof(float) * 84);
  hipMalloc((void**)&d_dense2_activated, sizeof(float) * 84);
  hipMalloc((void**)&d_dense3_weigths, sizeof(float) * 84 * 10);
  hipMalloc((void**)&d_dense3, sizeof(float) * 10);
  hipMalloc((void**)&d_pred, sizeof(float) * 10);


  // Layer 2: Première conv2D (6 kernels de taille 5x5 => matrice de taille 6x28x28)
  cudaConv2D<<<28,28>>>(d_raw_data, d_C1_kernel, d_C1_data, 32, 32, 5, 6, 28, 28);
  hipDeviceSynchronize();
  // Activation tanh
  activation_tanh<<<28,28>>>(d_C1_data, 28, 28, 6, d_C1_data_activated);
  hipDeviceSynchronize();

  // Layer 3: Sous échantillionnage par moyennage 2x2
  cudaMeanPool<<<14,14>>>(d_C1_data_activated, d_S1_data, 28, 28, 6, 2, 14, 14);
  hipDeviceSynchronize();

  // Layer 4: Deuxième conv2D (16 kernels de taille 5x5 => matrice de taille 16x10x10)
  cudaConv2D<<<14,14>>>(d_S1_data, d_C2_kernel, d_C2_data, 14, 14, 5, 16, 10, 10);
  hipDeviceSynchronize();
  // Activation tanh
  activation_tanh<<<10,10>>>(d_C2_data, 10, 10, 16, d_C2_data_activated);
  hipDeviceSynchronize();

  // Layer 5: Sous échantillionnage par moyennage 2x2
  cudaMeanPool<<<5,5>>>(d_C2_data_activated, d_S2_data, 10, 10, 16, 2, 5, 5);
  hipDeviceSynchronize();

  // Layer 6: Première couche Dense. size de 120
  Dense<<<120,1>>>(d_dense1_weigths, d_S2_data, d_dense1, 120);
  hipDeviceSynchronize();
  // Activation tanh
  activation_tanh<<<120,1>>>(d_dense1, 120, 1, 1, d_dense1_activated);
  hipDeviceSynchronize();

  // Layer 7: Deuxième couche Dense. size 84
  Dense<<<84,1>>>(d_dense2_weigths, d_dense1, d_dense2, 84);
  hipDeviceSynchronize();
  // Activation tanh
  activation_tanh<<<84,1>>>(d_dense2, 84, 1, 1, d_dense2_activated);
  hipDeviceSynchronize();

  // Layer 8: Troisième couche Dense. size 10
  Dense<<<10,1>>>(d_dense3_weigths, d_dense2, d_dense3, 10);
  hipDeviceSynchronize();
  // Activation softmax
  activation_softmax(d_dense3, 10, pred);
  hipDeviceSynchronize();

  // Copie du resultat GPU sur CPU
  hipMemcpy(pred, d_pred, sizeof(float) * 10, hipMemcpyDeviceToHost);

  printf("Prediction: %f \n", pred);

  hipFree(d_C1_data);
  hipFree(d_C1_data_activated);
  hipFree(d_S1_data);
  hipFree(d_C1_kernel);
  hipFree(d_raw_data);

  hipFree(d_C2_data);
  hipFree(d_C2_data_activated);
  hipFree(d_S2_data);
  hipFree(d_C2_kernel);

  free(C1_data);
  free(S1_data);
  free(raw_data);
  free(C1_kernel);

}