
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

void charBckgrndPrint(char *str, float rgb){
  printf("\033[48;2;%d;%d;%dm", (int) rgb*255, (int) rgb*255, (int) rgb*255);
  printf("%s\033[0m",str);
}

void imgColorPrint(int height, int width, float *img){
  int row, col;
  char *str="  ";
  for(row=0; row<height; row++){
    for(col=0; col<width; col++){
      charBckgrndPrint(str,img[row*width + col]);
    }
    printf("\n");
  }
}

void Matrix2DInitRand(float *M, int n, int p)
{
    srand(time(0));
    for(int i=0; i<n; i++) {
      for(int j=0;j<p;j++) {
         M[i*n + j] = (float)rand() / (float)RAND_MAX ;
      }
    }
}

void Matrix3DInitZero(float *M, int fm, int n, int p)
{
    for(int i=0; i<n; i++) {
      for(int j=0;j<p;j++) {
        for(int k=0;k<fm;k++) {
          M[i*p + j*fm + k] = 0;
        }
      }
    }
}

void Matrix3DInitRand(float *M, int fm, int n, int p)
{
  srand(time(0));
  for(int i=0; i<n; i++) {
    for(int j=0;j<p;j++) {
      for(int k=0;k<fm;k++) {
        M[i*p + j*fm + k] = (float)rand() / (float)RAND_MAX;
      }
    }
  }
}



void Matrix2DPrint(float *M, int n, int p)
{
    for(int i=0; i<n; i++) {
      for(int j=0;j<p;j++) {
          printf("%f |", M[i*n + j] );
      }
      printf ( "\n");
    }
}

__global__ void cudaConv2D(float* M, float* kernel, float* Mout, int M_ligne, int M_colonne, int kernel_size, int nb_kernel, int Mout_ligne, int Mout_colonne){
    
    //Convolution d'une matrice par un kernel
    int lig = blockIdx.x;
    int col = threadIdx.x;

    float s = 0.0;

    if (lig < Mout_ligne && col < Mout_colonne)
    {
        int tot = M_ligne * M_colonne;

        for (int kernel_lig = 0; kernel_lig < kernel_size; kernel_lig++) {
            for (int kernel_col = 0; kernel_col < kernel_size; kernel_col++) {
                for (int n_k = 0; n_k < nb_kernel; n_k++)
                {
                    s += M[(lig + kernel_lig) * M_colonne + col + kernel_col + n_k * tot] * kernel[kernel_lig * kernel_size + kernel_col + n_k * nb_kernel];
                }
            }
        }
        Mout[lig * Mout_colonne + col] = s;
    }
}

__global__ void cudaMeanPool(float* M, float* Mout, int M_ligne, int M_colonne, int M_prof, int meanpool_size, int Mout_ligne, int Mout_colonne){
    /*
      cudaMeanPool réalise un sous-échantillonanage par moyennage 2x2 pixels vers 1 pixel.
    */

    // On effectuera les calculs sur les lignes et colonnes paires. 
    // ATTENTION: Dans ce cas blockIdx.x et threadIdx.x vont de 0 à 13 donc lig et col de 0 à 26.
    int lig = 2*blockIdx.x;
    int col = 2*threadIdx.x;


        float s;
        int tot_meanpool = meanpool_size * meanpool_size;
        int tot_M = M_ligne * M_colonne;
        int tot_Mout = Mout_ligne * Mout_colonne;
        
        for (int n_prof = 0; n_prof < M_prof; n_prof++){
            s = 0.0;
            
            for (int meanpool_lig = 0; meanpool_lig < meanpool_size; meanpool_lig++) {
                for (int meanpool_col = 0; meanpool_col < meanpool_size; meanpool_col++) {
                    s += M[(lig + meanpool_lig) * M_colonne + col + meanpool_col + n_prof * tot_M] / tot_meanpool;
                }
            }

            Mout[blockIdx.x * Mout_colonne + threadIdx.x + n_prof * tot_Mout] = s;
            
        }
}

__global__ void activation_tanh(float* M, int M_ligne, int M_colonne, int M_prof, float* Mout){
    
    int lig = blockIdx.x;
    int col = threadIdx.x;

    int tot_M = M_ligne * M_colonne;
    
    for (int n_prof = 0; n_prof < M_prof; n_prof++){
        Mout[lig * M_colonne + col + n_prof * tot_M] = tanh(M[lig * M_colonne + col + n_prof * tot_M]);
    }

}




int main(int argc, char *argv[]){

  // Layer 1
  // Initialisation de la matrice d'entrée
  float *raw_data;
  raw_data = (float*)malloc(sizeof(float) * 32 * 32);
  Matrix2DInitRand(raw_data, 32, 32);

  //imgColorPrint(32, 32, raw_data);

  // Initialisation de la matrice de sortie de la première conv
  float *C1_data;
  C1_data = (float*)malloc(sizeof(float) * 6 * 28 * 28);
  Matrix3DInitZero(C1_data, 6, 28, 28);
   
  // Initialisation de la matrice de sortie du sous-echantillonnage
  float *S1_data;
  S1_data = (float*)malloc(sizeof(float) * 6 * 14 * 14);
  Matrix3DInitZero(S1_data, 6, 14, 14);
  
  // Initialisation de la matrice des premiers kernels
  float *C1_kernel;
  C1_kernel = (float*)malloc(sizeof(float) * 6 * 5 * 5);
  Matrix3DInitRand(C1_kernel, 6, 5, 5);

  // Copie des matrices dans la mémoire GPU afin d'effectuer les calculs du réseau
  
  float *d_raw_data, *d_C1_data, *d_C1_data_activated, *d_C1_kernel, *d_S1_data;

  hipMalloc((void**)&d_raw_data, sizeof(float) * 32 * 32 * 1);
  hipMemcpy(d_raw_data, raw_data, sizeof(float) * 32 * 32 * 1, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_C1_kernel, sizeof(float) * 5 * 5 * 6);
  hipMemcpy(d_C1_kernel, C1_kernel, sizeof(float) * 5 * 5 * 6, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_C1_data, sizeof(float) * 28 * 28 * 6);
  hipMemcpy(d_C1_data, C1_data, sizeof(float) * 28 * 28 * 6, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_C1_data_activated, sizeof(float) * 28 * 28 * 6);

  hipMalloc((void**)&d_S1_data, sizeof(float) * 14 * 14 * 6);

  // Layer 2: Première conv2D (6 kernels de taille 5x5 => matrice de taille 6x28x28)
  cudaConv2D<<<28,28>>>(d_raw_data, d_C1_kernel, d_C1_data, 32, 32, 5, 6, 28, 28);
  hipDeviceSynchronize();

  // Activation tanh
  activation_tanh<<<28,28>>>(d_C1_data, 28, 28, 6, d_C1_data_activated);
  hipDeviceSynchronize();

  // Layer 3: Sous échantillionnage par moyennage 2x2
  cudaMeanPool<<<14,14>>>(d_C1_data_activated, d_S1_data, 28, 28, 6, 2, 14, 14);
  hipDeviceSynchronize();

  // Copie du resultat GPU sur CPU
  hipMemcpy(S1_data, d_S1_data, sizeof(float) * 14 * 14 * 6, hipMemcpyDeviceToHost);
  
  // Un petit printf pour vérifier que la matrice S1 est non nulle
  printf("S1_data: \n");
  Matrix2DPrint(S1_data, 14, 14*6);

  hipFree(d_C1_data);
  hipFree(d_S1_data);
  hipFree(d_C1_kernel);
  hipFree(d_raw_data);

  free(C1_data);
  free(S1_data);
  free(raw_data);
  free(C1_kernel);

}