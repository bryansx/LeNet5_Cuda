
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

void Matrix2DInitRand(float *M, int n, int p)
{
    srand(time(0));
    for(int i=0; i<n; i++) {
      for(int j=0;j<p;j++) {
         M[i*n + j] = (float)rand() / (float)RAND_MAX ;
      }
    }
}

void Matrix3DInitZero(float *M, int fm, int n, int p)
{
    for(int i=0; i<n; i++) {
      for(int j=0;j<p;j++) {
        for(int k=0;j<fm;k++) {
          M[i*p + j*fm + k] = 0;
        }
      }
    }
}

void Matrix3DInitRand(float *M, int fm, int n, int p)
{
  srand(time(0));
  for(int i=0; i<n; i++) {
    for(int j=0;j<p;j++) {
      for(int k=0;j<fm;k++) {
        M[i*p + j*fm + k] = (float)rand() / (float)RAND_MAX;
      }
    }
  }
}



void Matrix2DPrint(float *M, int n, int p)
{
    for(int i=0; i<n; i++) {
      for(int j=0;j<p;j++) {
          printf("%f |", M[i*n + j] );
      }
      printf ( "\n");
    }
}

__global__ void cudaConv2D(float* M, float* kernel, float* Mout, int M_ligne, int M_colonne, int kernel_size, int nb_kernel, int Mout_ligne, int Mout_colonne){
    
    //Convolution d'une matrice par un kernel
    int lig = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float s = 0.0;

    if (lig < Mout_ligne && col < Mout_colonne)
    {
        int tot = M_ligne * M_colonne;

        for (int kernel_lig = 0; kernel_lig < kernel_size; kernel_lig++) {
            for (int kernel_col = 0; kernel_col < kernel_size; kernel_col++) {
                for (int n_k = 0; n_k < nb_kernel; n_k++)
                {
                    s += M[(lig + kernel_lig) * M_colonne + col + kernel_col + n_k * tot] * kernel[kernel_lig * kernel_size + kernel_col + n_k * nb_kernel];
            
                }
            }
        }
        Mout[lig * Mout_colonne + col] = s;
    }
}


int main(int argc, char *argv[]){

  // Layer 1

  // Initialisation de la matrice d'entrée
  float *raw_data;
  raw_data = (float*)malloc(sizeof(float) * 32 * 32);
  Matrix2DInitRand(raw_data, 32, 32);

  // Initialisation de la matrice de sortie de la première conv
  float *C1_data;
  C1_data = (float*)malloc(sizeof(float) * 6 * 28 * 28);
  Matrix3DInitZero(C1_data, 6, 28, 28);
   
  // Initialisation de la matrice de sortie du sous-echantillonnage
  float *S1_data;
  S1_data = (float*)malloc(sizeof(float) * 6 * 14 * 14);
  Matrix3DInitZero(S1_data, 6, 14, 14);
  
  // Initialisation de la matrice des premiers kernels
  float *C1_kernel;
  C1_kernel = (float*)malloc(sizeof(float) * 6 * 5 * 5);
  Matrix3DInitRand(C1_kernel, 6, 14, 14);

  // Layer 2



}