
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

void Matrix2DInitRand(float *M, int n, int p)
{
    srand(time(0));
    for(int i=0; i<n; i++) {
      for(int j=0;j<p;j++) {
         M[i*n + j] = (float)rand() / (float)RAND_MAX ;
      }
    }
}

void Matrix3DInitZero(float *M, int fm, int n, int p)
{
    for(int i=0; i<n; i++) {
      for(int j=0;j<p;j++) {
        for(int k=0;j<fm;k++) {
          M[i*p + j*fm + k] = 0;
        }
      }
    }
}

void Matrix3DInitRand(float *M, int fm, int n, int p)
{
  srand(time(0));
  for(int i=0; i<n; i++) {
    for(int j=0;j<p;j++) {
      for(int k=0;j<fm;k++) {
        M[i*p + j*fm + k] = (float)rand() / (float)RAND_MAX;
      }
    }
  }
}



void Matrix2DPrint(float *M, int n, int p)
{
    for(int i=0; i<n; i++) {
      for(int j=0;j<p;j++) {
          printf("%f |", M[i*n + j] );
      }
      printf ( "\n");
    }
}



int main(int argc, char *argv[]){

  // Initialisation de la matrice d'entrée
  float *raw_data;
  raw_data = (float*)malloc(sizeof(float) * 32 * 32);
  Matrix2DInitRand(raw_data, 32, 32);

  // Initialisation de la matrice de sortie de la première conv
  float *C1_data;
  C1_data = (float*)malloc(sizeof(float) * 6 * 28 * 28);
  Matrix3DInitZero(C1_data, 6, 28, 28);
   
  // Initialisation de la matrice de sortie du sous-echantillonnage
  float *S1_data;
  S1_data = (float*)malloc(sizeof(float) * 6 * 14 * 14);
  Matrix3DInitZero(S1_data, 6, 14, 14);
  
  // Initialisation de la matrice des premiers kernels
  float *C1_kernel;
  C1_kernel = (float*)malloc(sizeof(float) * 6 * 5 * 5);
  Matrix3DInitRand(C1_kernel, 6, 14, 14);

}