
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

void charBckgrndPrint(char *str, float rgb){
  printf("\033[48;2;%d;%d;%dm", (int) rgb*255, (int) rgb*255, (int) rgb*255);
  printf("%s\033[0m",str);
}

void imgColorPrint(int height, int width, float *img){
  int row, col;
  char *str="  ";
  for(row=0; row<height; row++){
    for(col=0; col<width; col++){
      charBckgrndPrint(str,img[row*width + col]);
    }
    printf("\n");
  }
}

void Matrix2DInitRand(float *M, int n, int p)
{
    srand(time(0));
    for(int i=0; i<n; i++) {
      for(int j=0;j<p;j++) {
         M[i*n + j] = (float)rand() / (float)RAND_MAX ;
      }
    }
}

void Matrix3DInitZero(float *M, int fm, int n, int p)
{
    for(int i=0; i<n; i++) {
      for(int j=0;j<p;j++) {
        for(int k=0;j<fm;k++) {
          M[i*p + j*fm + k] = 0;
        }
      }
    }
}

void Matrix3DInitRand(float *M, int fm, int n, int p)
{
  srand(time(0));
  for(int i=0; i<n; i++) {
    for(int j=0;j<p;j++) {
      for(int k=0;j<fm;k++) {
        M[i*p + j*fm + k] = (float)rand() / (float)RAND_MAX;
      }
    }
  }
}



void Matrix2DPrint(float *M, int n, int p)
{
    for(int i=0; i<n; i++) {
      for(int j=0;j<p;j++) {
          printf("%f |", M[i*n + j] );
      }
      printf ( "\n");
    }
}

__global__ void cudaConv2D(float* M, float* kernel, float* Mout, int M_ligne, int M_colonne, int kernel_size, int nb_kernel, int Mout_ligne, int Mout_colonne){
    
    //Convolution d'une matrice par un kernel
    int lig = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float s = 0.0;

    if (lig < Mout_ligne && col < Mout_colonne)
    {
        int tot = M_ligne * M_colonne;

        for (int kernel_lig = 0; kernel_lig < kernel_size; kernel_lig++) {
            for (int kernel_col = 0; kernel_col < kernel_size; kernel_col++) {
                for (int n_k = 0; n_k < nb_kernel; n_k++)
                {
                    s += M[(lig + kernel_lig) * M_colonne + col + kernel_col + n_k * tot] * kernel[kernel_lig * kernel_size + kernel_col + n_k * nb_kernel];
            
                }
            }
        }
        Mout[lig * Mout_colonne + col] = s;
    }
}


int main(int argc, char *argv[]){

  // Layer 1
  // Initialisation de la matrice d'entrée
  float *raw_data;
  raw_data = (float*)malloc(sizeof(float) * 32 * 32);
  Matrix2DInitRand(raw_data, 32, 32);

  imgColorPrint(32, 32, raw_data);

  // Initialisation de la matrice de sortie de la première conv
  float *C1_data;
  C1_data = (float*)malloc(sizeof(float) * 6 * 28 * 28);
  Matrix3DInitZero(C1_data, 6, 28, 28);
   
  // Initialisation de la matrice de sortie du sous-echantillonnage
  float *S1_data;
  S1_data = (float*)malloc(sizeof(float) * 6 * 14 * 14);
  Matrix3DInitZero(S1_data, 6, 14, 14);
  
  // Initialisation de la matrice des premiers kernels
  float *C1_kernel;
  C1_kernel = (float*)malloc(sizeof(float) * 6 * 5 * 5);
  Matrix3DInitRand(C1_kernel, 6, 14, 14);

  // Layer 2
  // Premiere convolution
  float *d_raw_data, *d_C1_data, *d_C1_kernel, *d_S1_data;

  hipMalloc((void**)&d_raw_data, sizeof(float) * 32 * 32 * 1);
  hipMemcpy(d_raw_data, raw_data, sizeof(float) * 32 * 32 * 1, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_C1_kernel, sizeof(float) * 5 * 5 * 6);
  hipMemcpy(d_C1_kernel, C1_kernel, sizeof(float) * 5 * 5 * 6, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_C1_data, sizeof(float) * 28 * 28 * 6);
  hipMemcpy(d_C1_data, C1_data, sizeof(float) * 28 * 28 * 6, hipMemcpyHostToDevice);

  cudaConv2D<<<32,32>>>(d_raw_data, d_C1_kernel, d_C1_data, 32, 32, 5, 6, 28, 28);
  hipDeviceSynchronize();

  // Copie du resultat sur GPU
  hipMemcpy(C1_data, d_C1_data, sizeof(float) * 28 * 28 * 6, hipMemcpyDeviceToHost);
  //imgColorPrint(28, 28, C1_data);
}